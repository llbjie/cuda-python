#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <thrust/device_ptr.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>

#include <iostream>
// #include <hip/hip_runtime_api.h>

/////////////////////////////////////////////////////////////////
// Some utility code to define grid_stride_range
// Normally this would be in a header but it's here
// for didactic purposes. Uses
#include "range.hpp"
using namespace util::lang;


// type alias to simplify typing...
template <typename T>
using step_range = typename range_proxy<T>::step_range_proxy;

template <typename T>
__device__ step_range<T> grid_stride_range(T begin, T end) {
  begin += blockDim.x * blockIdx.x + threadIdx.x;
  return range(begin, end).step(gridDim.x * blockDim.x);
}
/////////////////////////////////////////////////////////////////

template <typename T, typename Predicate>
__device__ void count_if(int *count, T *data, int n, Predicate p) {
  for (auto i : grid_stride_range(0, n)) {
    if (p(data[i])) atomicAdd(count, 1);
  }
}

extern "C" {
// Use count_if with a lambda function that searches for x, y, z or w
// Note the use of range-based for loop and initializer_list inside the functor
// We use auto so we don't have to know the type of the functor or array
__global__ void xyzw_frequency(int *count, char *text, int n) {
  const char letters[]{'x', 'y', 'z', 'w'};

  count_if(count, text, n, [&](char c) {
    for (const auto x : letters)
      if (c == x) return true;
    return false;
  });
}

__global__ void xyzw_frequency_thrust_device(int *count, char *text, int n) {
  const char letters[]{'x', 'y', 'z', 'w'};
  *count = thrust::count_if(thrust::device, text, text + n, [=](char c) {
    for (const auto x : letters)
      if (c == x) return true;
    return false;
  });
}

// a bug in Thrust 1.8 causes warnings when this is uncommented
// so commented out by default -- fixed in Thrust master branch
#if 0 
void xyzw_frequency_thrust_host(int *count, char *text, int n)
{
  const char letters[] { 'x','y','z','w' };
  *count = thrust::count_if(thrust::host, text, text+n, [&](char c) {
    for (const auto x : letters) 
      if (c == x) return true;
    return false;
  });
}
#endif
}
